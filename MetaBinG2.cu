#include "hip/hip_runtime.h"
/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
/* Includes, cuda */
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include <sys/timeb.h>
#include <pthread.h>

//global variable
int *statsNum;
char** tmp;
char** tmp1;

int threadNum;


int kmer=0;
int seqvlen;
int buff;
int mrow;
float alpha;

char **datas;
float* datad;
// float* d_datad;
float* sigmaG;
//float *d_sigmaG;
int flag;
int mcolmax;

long long getSystemTime(){
        struct timeb t;
        ftime(&t);
        return 1000*t.time+t.millitm;
}



void checkCUDAError(const char *msg)     
{     
    hipError_t err = hipGetLastError();     
    if( hipSuccess != err)      
    {     
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,      
        hipGetErrorString( err) );     
        exit(EXIT_FAILURE);     
    }                              
}    
void classify(char** datas,float* scorelist,int buff,int mrow,char** titlel,float* sigma,int* statsNum,int flag,float alpha){
     for(int i=0;i<buff;i++){
         float s1=1e10;
         int r1=0;
         float stmp=0;

         for(int j=1;j<mrow;j++){
             stmp=scorelist[j*buff+i]*(1+alpha*sigma[j]);
             if(stmp<s1){
                 s1=stmp;r1=j*buff+i;
             }
         }

         statsNum[(r1-i)/buff]++;


         if(flag){
             printf("%s",titlel[i]);
             for(int j=6;j>=0;j--){
                 printf("\t%s",datas[7*(r1-i)/buff+j]);
             }
             printf("\n");
         }

     }
	
}
//cuda kernel
__global__ void parallel(float *d_scorelist,float *d_sigma,int *d_index,int buff,int mrow,float alpha){
        int i=threadIdx.x+blockIdx.x*blockDim.x;
        int j;
        float m=1e10;
        float tmp;
        if(i<buff){
				d_index[i]=0;
                for(j=0;j<mrow;j++){
                        tmp=d_scorelist[j*buff+i]*(1+d_sigma[j]*alpha);
                        if(tmp<m){
                                m=tmp;
                                d_index[i]=j;
                        }
                }
        }
		__syncthreads();
}

//======================================================================================
//usage
//======================================================================================
void usage(){
	printf("Usage: ./MetaBinG2 [FASTA file] [db] [threadNum] [outname]\n");
}

//======================================================================================
//calculate the index of kmer 
//parameters
//str:kmer
//======================================================================================
int calKmerIndex(char* str){
	int a=0;
	int n=strlen(str);
	int tmp=0;
	//order A T C G
	for(int i=0;i<n;i++){
		switch(str[i]){
			case 'A':
				tmp=0;
				break;
			case 'a':
				tmp=0;
				break;	
			case 'T':
				tmp=1;
				break;
			case 't':
				tmp=1;
				break;
			case 'C':
				tmp=2;
				break;
			case 'c':
				tmp=2;
				break;
			case 'G':
				tmp=3;
				break;
			case 'g':
				tmp=3;
				break;
			default:
				;	
		}
		a=a*4+tmp;
	}
	
	return a;
}

//======================================================================================
//count kmer of a seq
//parameters: 
//seq: atcg sequences
//kmer: size of kmer
//frag: a temp variant used to store kmers
//======================================================================================
void countKmer(char* seq,int kmer,char* frag,float* seqm,int buffi,int seqvlen){
	for(int i=0;i<strlen(seq)-kmer+1;i++){
		int legal=1;
		for(int j=0;j<kmer;j++){
			if(seq[i+j]!='A' && seq[i+j]!='T' && seq[i+j]!='C' && seq[i+j]!='G' && seq[i+j]!='a' && seq[i+j]!='t' && seq[i+j]!='c' && seq[i+j]!='g'){
				legal=0;
			}
		}
		if(legal==1){
	                strncpy(frag,seq+i,kmer);
			seqm[calKmerIndex(frag)+buffi*seqvlen]+=1;
		}
	}
}

//======================================================================================
//do matrix multiplication on GPU
//parameters
//scorelist:mrow*buff matrix
//d_scorelist:gpu scorelist
//seqm:buff*seqvlen matrix
//d_seqm:gpu seqm
//d_datad:  the numberic part of the db file,mrow*seqvlen matrix
//======================================================================================

int matMultiplication(int mrow,int buff,float* scorelist,float* d_scorelist,int seqvlen,float* seqm,float* d_seqm,float* d_datad, char ** datas,char** titlel,float* sigmaG,int* statsNum,int flag,float alpha,float* d_sigmaG,int* d_index,int* index,int buffi,FILE *fp){
	hipblasStatus_t status;

    status=hipblasSetVector(mrow*buff, sizeof(float), scorelist, 1, d_scorelist, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
            fprintf (stderr, "device access error.\n");
        return EXIT_FAILURE;
    }

    status=hipblasSetVector(seqvlen*buff, sizeof(float), seqm, 1, d_seqm, 1);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf (stderr, "device access error.\n");
        return EXIT_FAILURE;
    }

    hipblasSgemm('t', 'n', buff, mrow, seqvlen, 1.0f, d_seqm, seqvlen, d_datad, seqvlen, 0.0, d_scorelist, buff);

    int readNum;

	hipMemcpy(d_sigmaG,sigmaG,sizeof(float)*mrow,hipMemcpyHostToDevice);
	
	dim3 grid(ceil((buff+127)/128),1,1);
	dim3 block(128,1,1);
	parallel<<<grid,block>>>(d_scorelist,d_sigmaG,d_index,buff,mrow,alpha);
	
	hipMemcpy(index,d_index,sizeof(int)*buff,hipMemcpyDeviceToHost);

	if(buffi==0){
		readNum=buff;
	}else{
		readNum=buffi;
	}	

	for(int i=0;i<readNum;i++){
        statsNum[index[i]]++;    
		if(flag){
            fprintf(fp,"%s",titlel[i]);
            for(int j=6;j>=0;j--){
                fprintf(fp,"\t%s",datas[7*index[i]+j]);
            }
            fprintf(fp,"\n");
        }
		
    }
	
    return 0;
}

void *classFun(void *arg){
	int ci=*(int *)arg;

    int i,j;
    char* line;
    line=(char*)malloc((10000+(kmer+2))*seqvlen*sizeof(char));
    memset(line,0,(10000+(kmer+2)*seqvlen));

    int buffi=0;

    char* seq=(char*)malloc((10000+(kmer+2))*seqvlen*sizeof(char));
    float* seqm=(float*)malloc(sizeof(float)*seqvlen*buff);
    for(int i=0;i<seqvlen*buff;i++){
			seqm[i]=0;
	}
	float* d_seqm=0;
	cublasAlloc(seqvlen*buff,sizeof(float),(void**)&d_seqm);

	char* title=(char*)malloc(sizeof(char)*1000);
	char** titlel=(char**)malloc(buff*sizeof(char*));
	for(int i=0;i<buff;i++){
		titlel[i]=(char*)malloc((1000)*sizeof(char));
	}
	char* frag;
	frag=(char*)malloc(sizeof(char)*10);
	float* scorelist=(float*)malloc(buff*mrow*sizeof(float));
	float* d_scorelist = 0;
	cublasAlloc(mrow*buff, sizeof(float),(void**)&d_scorelist);		

	int *index=(int*)malloc(sizeof(int)*buff);
	int *d_index;
	hipMalloc((void**)&d_index,sizeof(int)*buff);

	int *statsNumthread=(int *)malloc(mrow*sizeof(int));

	for(i=0;i<mrow;i++){
		statsNumthread[i]=0;
	}

	char** datas_fun=(char**)malloc(7*mrow*sizeof(char*));
	for(i=0;i<mrow;i++){
		for(j=0;j<7;j++){
			datas_fun[i*7+j]=(char*)malloc((strlen(datas[i*7+j])+1)*sizeof(char));
			strcpy(datas_fun[i*7+j],datas[i*7+j]);
		}
	}

	float* datad_fun=(float*)malloc((mcolmax-7)*mrow*sizeof(float));
	memcpy(datad_fun,datad,sizeof(float)*mrow*(mcolmax-7));
	float* d_datad=0;
	cublasAlloc(mrow*(mcolmax-7), sizeof(float),(void**)&d_datad);
	hipblasSetVector(mrow*(mcolmax-7), sizeof(float), datad_fun, 1, d_datad, 1);

	float *sigmaG_fun=(float*)malloc(mrow*sizeof(float));
	memcpy(sigmaG_fun,sigmaG,sizeof(float)*mrow);
	float *d_sigmaG;
	hipMalloc((void**)&d_sigmaG,sizeof(float)*mrow);


	FILE *fp=fopen(tmp[ci],"r");
	FILE *fp1=fopen(tmp1[ci],"w+");

    while(fgets(line,(10000+(kmer+2)*seqvlen),fp)!=NULL){
            line[strlen(line)-1]='\0';
            if(line[0]=='>'){
					if(title[0]=='>'){
						countKmer(seq,kmer,frag,seqm,buffi,seqvlen);
						strcpy(titlel[buffi],title);
						
						buffi++;
						if(buffi==buff){
							buffi=0;
							for(int i=0;i<buff*mrow;i++){
								scorelist[i]=0;
							}				   
							matMultiplication(mrow,buff,scorelist,d_scorelist,seqvlen,seqm,d_seqm,d_datad,datas_fun,titlel,sigmaG_fun,statsNumthread,flag,alpha,d_sigmaG,d_index,index,buffi,fp1);
							for(int i=0;i<seqvlen*buff;i++){
								seqm[i]=0;
							}
						}
					}
					memset(title,0,1000);
					strcpy(title,line);
					memset(seq,0,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))));
					memset(line,0,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))));			
				}else{
					strcpy(seq,strcat(seq,line));
				}

    }

    fclose(fp);
    if(title[0]=='>'){
		countKmer(seq,kmer,frag,seqm,buffi,seqvlen);
		strcpy(titlel[buffi],title);
		buffi++;
		for(int i=0;i<buff*mrow;i++){
					scorelist[i]=0;
		}

		matMultiplication(mrow,buff,scorelist,d_scorelist,seqvlen,seqm,d_seqm,d_datad,datas_fun,titlel,sigmaG_fun,statsNumthread,flag,alpha,d_sigmaG,d_index,index,buffi,fp1);
		memset(title,0,1000);
		memset(seq,0,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))));
		memset(line,0,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))));
	}

    for(i=0;i<mrow;i++){
            statsNum[ci*mrow+i]=statsNumthread[i];
    }

    fclose(fp1);

    free(index);				
	hipFree(d_index);
	hipFree(d_sigmaG);
	hipFree(d_datad);
	for(int i=0;i<buff;i++){free(titlel[i]);}
	for(int i=0;i<7*mrow;i++){free(datas_fun[i]);}
	free(line);			
	free(title);			
	free(seq);
	cublasFree(d_seqm);
	free(frag);
	free(scorelist);
	cublasFree(d_scorelist);
	free(seqm);


    return ((void *)0);	
}

int main(int argc,char ** argv){
	if(argc==5){
		long long start=getSystemTime();

		hipblasStatus_t status;
		status = cublasInit();
		if (status != HIPBLAS_STATUS_SUCCESS) {
				fprintf (stderr, "CUBLAS initialization error.\n");
	        	return EXIT_FAILURE;
	    	}
		int mcol=0;
		mrow=0;
		mcolmax=0;
		buff=1000;
		//alpha=atof(argv[3]);
		alpha=0.0002;
		threadNum=atoi(argv[3]);

		//if(!alpha>0){
			//alpha=0.0002;
		//}
		if(!threadNum>0){
			threadNum=4;
		}else if(threadNum>16){
			threadNum=4;
		}

		printf("You selected %d threads.\n",threadNum);

		tmp=(char**)malloc(threadNum*sizeof(char*));
		tmp1=(char**)malloc(threadNum*sizeof(char*));

		for(int i=0;i<threadNum;i++){
			tmp[i]=(char*)malloc(100*sizeof(char));
			tmp1[i]=(char*)malloc(100*sizeof(char));
			sprintf(tmp[i],"%s_tmp_%d",argv[4],i);
			sprintf(tmp1[i],"%s_tmpw_%d",argv[4],i);
		}


		//======================================================================================
		//read the dbfile to determine the number of rows and cols
		//======================================================================================
		FILE *fp1;
		char ch;
		const char* db=NULL;
		db=argv[2];
		//if(argc==2){db="db";}
		fp1=fopen(db,"r");
		if(fp1==NULL){
			printf("The db file does not exist!\n");exit(1);
		}
		while((ch=fgetc(fp1))!=EOF){
			switch(ch){
				case '\t':
					mcol++;
					break;
				case '\n':
					mrow++;
					if(mcol>mcolmax){
						mcolmax=mcol;
					}
					mcol=0;
					break;
				default:
					break;
			}		
		}
		mcolmax++;
		kmer=0;
		int ncoltemp=mcolmax-7;
		while(ncoltemp>=4){
			ncoltemp=ncoltemp/4;
			kmer++;
		}
		if(ncoltemp!=1){
			printf("%s","The db file format is not correct!\n");
			exit(1);
		}
		seqvlen=(unsigned int)pow(4.0,kmer);
		fclose(fp1);

		//======================================================================================
		//read the dbfile again to get the taxonomy information and the transition probabilities
		//======================================================================================
		datas=(char**)malloc(7*mrow*sizeof(char*));

		datad=(float*)malloc((mcolmax-7)*mrow*sizeof(float));

		FILE *refp1;
		refp1=fopen(db,"r");
		char* line;
		int trow=0;
		int tcol=0;
		line=(char*)malloc((10000+(kmer+2)*((unsigned int)pow(4.0,kmer)))*sizeof(char));
		memset(line,0,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))));
		while(fgets(line,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))),refp1)!=NULL){
			line[strlen(line)-1]='\0';
			char* p;
			for(p=strtok(line,"\t\n");p;p=strtok(NULL,"\t\n")){
				if(tcol<7){
					datas[trow*7+tcol]=(char*)malloc((strlen(p)+1)*sizeof(char));
					strcpy(datas[trow*7+tcol],p);
				}else{
					datad[trow*(mcolmax-7)+tcol-7]=atof(p);
				}
				tcol++;
				if(tcol==mcolmax){
					trow++;
					tcol=0;
				}
			}
			memset(line,0,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))));
		}
		fclose(refp1);

		//======================================================================================
		//divide the sequence file into 4 tmp file
		//======================================================================================
		FILE *fseq;

		FILE *fseqt[16];

        fseq=fopen(argv[1],"r");

        for(int i=0;i<threadNum;i++){
        	fseqt[i]=fopen(tmp[i],"w+");
        }

        
        memset(line,0,(10000+(kmer+2)*((unsigned int)pow(4.0,kmer))));
        long int readNum=0;
        while(fgets(line,(10000+(kmer+2)*4096),fseq)!=NULL){
                readNum++;
        }

        fclose(fseq);

		long int tnum=readNum/threadNum+1;
		if(tnum%2!=0){
			tnum++;
		}

		fseq=fopen(argv[1],"r");

		long int num=0;

		while(fgets(line,(10000+(kmer+2)*4096),fseq)!=NULL){
                num++;
                	line[strlen(line)-1]='\0';
			 		if(num<=tnum){
	                    fprintf(fseqt[0],"%s\n",line);
	                }
	                else if(num<=tnum*2){
	                    fprintf(fseqt[1],"%s\n",line);
	                }else if(num<=tnum*3){
	                 	fprintf(fseqt[2],"%s\n",line);
	                }else if(num<=tnum*4){
	                 	fprintf(fseqt[3],"%s\n",line);
	                }else if(num<=tnum*5){
	                 	fprintf(fseqt[4],"%s\n",line);
	                }else if(num<=tnum*6){
	                 	fprintf(fseqt[5],"%s\n",line);
	                }else if(num<=tnum*7){
	                 	fprintf(fseqt[6],"%s\n",line);
	                }else if(num<=tnum*8){
	                 	fprintf(fseqt[7],"%s\n",line);
	                }else if(num<=tnum*9){
	                 	fprintf(fseqt[8],"%s\n",line);
	                }else if(num<=tnum*10){
	                 	fprintf(fseqt[9],"%s\n",line);
	                }else if(num<=tnum*11){
	                 	fprintf(fseqt[10],"%s\n",line);
	                }else if(num<=tnum*12){
	                 	fprintf(fseqt[11],"%s\n",line);
	                }else if(num<=tnum*13){
	                 	fprintf(fseqt[12],"%s\n",line);
	                }else if(num<=tnum*14){
	                 	fprintf(fseqt[13],"%s\n",line);
	                }else if(num<=tnum*15){
	                 	fprintf(fseqt[14],"%s\n",line);
	                }else{
	                 	fprintf(fseqt[15],"%s\n",line);
	                }
                
		
        }

        fclose(fseq);

        for(int i=0;i<threadNum;i++){
        	fclose(fseqt[i]);
        }

	    sigmaG=(float *)malloc(mrow*sizeof(float));
	    float *percentl=(float *)malloc(mrow*sizeof(float));

		for(int i=0;i<mrow;i++){
			sigmaG[i]=(float)mrow;
			percentl[i]=10;
		}

		//set cutoff
		float stop=0.1;

		flag=0;

		statsNum=(int *)malloc(mrow*threadNum*sizeof(int));


	    for(int t=0;t<10;t++){

	    	for(int i=0;i<mrow*threadNum;i++){
				statsNum[i]=0;
			}

			float BC=0;

	    	if(t>=9){
	    		flag=1;
	    	}
			
			//======================================================================================
			//1.read (buff) fasta sequences 
			//2.convert them into numeric vectors 
			//3.copy the vectors to GPU
			//4.do matrix multiplication with d_datad on GPU
			//5.get the result from the GPU
			//6.interpret the result
			//======================================================================================

	    	//multi-threads for reading fasta file
	    	pthread_t thread[16];
	    	int si[16]={0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15};
	        memset(&thread,0,sizeof(thread));

	        int i;

	        for(i=0;i<threadNum;i++){
	                int ci=si[i];
	                if(pthread_create(&thread[ci],NULL,&classFun,&si[i])!=0){
	                        printf("create thread error!\n");
	                }
	        }
	        int counter=0;
	        int wait=0;
	        int ti;

	        while(counter<threadNum){
	                wait++;

	                for(ti=0;ti<threadNum;ti++){
	                        if(thread[ti]!=0){
	                                pthread_join(thread[ti],NULL);
	                                counter++;
	                        }
	                }
	        }

	        

	        counter=0;

			if(flag){
				break;
			}else{
				//update sigma with statsNum
				int statsNumsum;

				int readscount=readNum/2;
				for(int i=0;i<mrow;i++){
					statsNumsum=0;
					for(int j=0;j<threadNum;j++){
						statsNumsum=statsNumsum+statsNum[j*mrow+i];
					}
					if(statsNumsum>0){
						
						sigmaG[i]=(float)readscount/(float)statsNumsum;
						BC=BC+pow(((float)statsNumsum/(float)readscount-percentl[i]),2);
						percentl[i]=(float)statsNumsum/(float)readscount;
					}else{
						sigmaG[i]=1e10;
						BC=BC+pow((0-percentl[i]),2);
						percentl[i]=0;
					}
				}
				printf("j:%d;BC:%.4f\n",j,BC);
				if(BC<stop){
					flag=1;
				}

			}
			
		}

		free(sigmaG);
		free(percentl);
		free(statsNum);
		for(int i=0;i<7*mrow;i++){free(datas[i]);}			
		free(datad);		

		char cmd[100];
		sprintf(cmd,"cat %s_tmpw_* >%s",argv[4],argv[4]);
		system(cmd);

		char cmd1[100];
		sprintf(cmd1,"rm %s_tmp*",argv[4]);
		system(cmd1);


		long long end=getSystemTime();
        printf("time:%lld ms\n",end-start);

		return 0;
	}else{
		usage();
		return -1;
	}
}
